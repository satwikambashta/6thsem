#include "hip/hip_runtime.h"
/* count number of times a given word is repeated in a sentence*/

##include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDACount(char *A, unsigned int *d_count, char *word, int wordLength)
{
    int i = threadIdx.x;
    int j;
    for (j = 0; j < wordLength; j++) {
        if (A[i + j] != word[j]) {
            break;
        }
    }
    if (j == wordLength) {
        atomicAdd(d_count, 1);
    }
}

int main()
{
    char A[N];
    char *d_A;
    unsigned int count = 0, *d_count;
    char word[N]; // Buffer to store the word to search for

    printf("Enter a string: ");
    if (scanf("%99[^\n\t]", A) != 1) {
        printf("Failed to read the string.\n");
        return 1; // Return an error code
    }

    // Clear the input buffer to remove the newline character left by the previous scanf
    int c;
    while ((c = getchar()) != '\n' && c != EOF);

    printf("Enter a word to search for: ");
    if (scanf(" %99[^\n\t]", word) != 1) { // Note the space before the format specifier
        printf("Failed to read the word.\n");
        return 1; // Return an error code
    }

    int wordLength = strlen(word);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void **)&d_A, strlen(A) * sizeof(char));
    hipMalloc((void **)&d_count, sizeof(unsigned int));

    hipMemcpy(d_A, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice); // Pass the address of count

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA Error1: %s\n", hipGetErrorString(error));
    }

    // Allocate memory for the word on the device
    char *d_word;
    hipMalloc((void **)&d_word, wordLength * sizeof(char));
    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);

    // Launch the kernel with the word and its length
    CUDACount<<<1, strlen(A)>>>(d_A, d_count, d_word, wordLength);

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA Error2: %s\n", hipGetErrorString(error));
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(&count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost); // Copy back the result

    printf("Total occurrences of '%s' = %d\n", word, count);
    printf("Time Taken = %f ms\n", elapsedTime);

    hipFree(d_A);
    hipFree(d_count);
    hipFree(d_word); // Free the memory allocated for the word on the device

    return 0;
}
