/* write a cuda program to read string s and produce string rs.
sample input S: PCAP .sample output RS:PCAPPCAPCP*/
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void GenerateRS(char *S, char *RS, int length)
{
    int i = threadIdx.x;
    for (int j = 0; j < length; j++)
    {
        RS[i * length + j] = S[i];
    }
}

int main()
{
    char S[N] = "PCAP";
    char RS[N * N]; // Assuming maximum length for RS

    int length = strlen(S);

    hipMalloc((void **)&RS, length * length * sizeof(char));

    GenerateRS<<<1, length>>>(S, RS, length);

    hipMemcpy(RS, RS, length * length * sizeof(char), hipMemcpyDeviceToHost);

    printf("Input string S: %s\n", S);
    printf("Output string RS: %s\n", RS);

    hipFree(RS);

    return 0;
}
