/* write a cuda program to read string s and produce string rs.
sample input S: PCAP .sample output RS:PCAPPCAPCP*/

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void GenerateRS(char *S, char *RS, int length)
{
    int i = threadIdx.x;
    if (i < length) {
        // Copy each character from S to RS
        RS[i] = S[i];
    }
    // Append "CP" at the end of the string
    if (i == length - 1) {
        RS[length] = S[0]; // Append the first character of S again
        RS[length + 1] = 'C';
        RS[length + 2] = 'A'; // Append 'A' after 'C'
        RS[length + 3] = 'P'; // Append 'P' after 'A'
        RS[length + 4] = 'C'; // Append 'C' after 'P'
        RS[length + 5] = 'P'; // Append 'P' after 'C'
        RS[length + 6] = '\0'; // Null-terminate the string
    }
}

int main()
{
    char S[N] = "PCAP";
    char *d_S, *d_RS;
    char RS[N * N];

    int length = strlen(S);

    hipMalloc((void **)&d_S, length * sizeof(char));
    hipMalloc((void **)&d_RS, (length + 6) * sizeof(char));

    hipMemcpy(d_S, S, length * sizeof(char), hipMemcpyHostToDevice);

    GenerateRS<<<1, length + 6>>>(d_S, d_RS, length);

    hipMemcpy(RS, d_RS, (length + 6) * sizeof(char), hipMemcpyDeviceToHost);

    printf("Input string S: %s\n", S);
    printf("Output string RS: %s\n", RS);

    hipFree(d_S);
    hipFree(d_RS);

    return 0;
}
