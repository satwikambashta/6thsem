/*
Write a CUDA program that reads a matrix A of size MXN and produce an output matrix B of same size such that
it replaces all the non-border elements(numbers in bold) of A with its equivalent 1's complement and remaining elements same as matrix A.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void modifyMatrix(int *A, int *B, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int idx = row * N + col;
        if (row > 0 && row < M - 1 && col > 0 && col < N - 1) {
            B[idx] = ~A[idx]; // 1's complement for non-border elements
        } else {
            B[idx] = A[idx]; // Same as matrix A for border elements
        }
    }
}

int main() {
    int M, N;
    printf("Enter the number of rows (M): ");
    scanf("%d", &M);
    printf("Enter the number of columns (N): ");
    scanf("%d", &N);
    int *A = (int *)malloc(M * N * sizeof(int));
    int *B = (int *)malloc(M * N * sizeof(int));

    // Input the matrix
    printf("Enter the matrix elements:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%d", &A[i * N + j]);
        }
    }

    // Allocate device memory
    int *d_A, *d_B;
    hipMalloc((void **)&d_A, M * N * sizeof(int));
    hipMalloc((void **)&d_B, M * N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);

    // Perform the operation
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    modifyMatrix<<<numBlocks, threadsPerBlock>>>(d_A, d_B, M, N);

    // Copy the result back to the host
    hipMemcpy(B, d_B, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    printf("Modified matrix B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", B[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    free(A);
    free(B);

    return 0;
}
