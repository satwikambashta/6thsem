/* Write a program in CUDA to read MXN matrix A and replace 1st row of this matrix by same elements,
 2nd row elements by square of each element and 3rd row elements by cube of each element and so on. */

 #include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void modifyMatrix(float *A, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int idx = row * N + col;
        float power = row + 1;
        A[idx] = pow(A[idx], power);
    }
}

int main() {
    int M, N;
    printf("Enter the number of rows (M): ");
    scanf("%d", &M);
    printf("Enter the number of columns (N): ");
    scanf("%d", &N);

    // Allocate memory for the matrix
    float *A = (float *)malloc(M * N * sizeof(float));

    // Input the matrix
    printf("Enter the matrix elements:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%f", &A[i * N + j]);
        }
    }

    // Allocate device memory
    float *d_A;
    hipMalloc((void **)&d_A, M * N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice);

    // Perform the operation
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    modifyMatrix<<<numBlocks, threadsPerBlock>>>(d_A, M, N);

    // Copy the result back to the host
    hipMemcpy(A, d_A, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    printf("Modified matrix:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", A[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    free(A);

    return 0;
}
