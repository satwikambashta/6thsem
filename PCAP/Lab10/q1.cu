/*Write a program in CUDA to perform parallel Sparse Matrix Vector multiplication using 
compressed sparse row (CSR) storage format.
Represent the input sparse matrix in CSR format in the host code.*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void spmv_csr(const int *csrRowPtrA, const int *csrColIndA, const float *csrValA, const float *x, float *y, int m) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        
        float sum = 0.0f;
        for (int i = csrRowPtrA[row]; i < csrRowPtrA[row + 1]; i++) {
            sum += csrValA[i] * x[csrColIndA[i]];
        }
        y[row] = sum;
    }
}

int main() {
    int m, n, nnz;
    printf("Enter the number of rows (m): ");
    scanf("%d", &m);
    printf("Enter the number of columns (n): ");
    scanf("%d", &n);
    printf("Enter the number of non-zero elements (nnz): ");
    scanf("%d", &nnz);

    // Allocate memory for the original matrix and the CSR format
    float *A = (float *)malloc(m * n * sizeof(float));
    int *csrRowPtrA = (int *)malloc((m + 1) * sizeof(int));
    int *csrColIndA = (int *)malloc(nnz * sizeof(int));
    float *csrValA = (float *)malloc(nnz * sizeof(float));

    // Input the original matrix
    printf("Enter the matrix elements:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            scanf("%f", &A[i * n + j]);
            if (A[i * n + j] != 0.0f) {
                csrValA[csrRowPtrA[i]] = A[i * n + j];
                csrColIndA[csrRowPtrA[i]] = j;
                csrRowPtrA[i + 1] = csrRowPtrA[i] + 1;
            }
        }
    }

    // Allocate device memory
    int *d_csrRowPtrA, *d_csrColIndA;
    float *d_csrValA, *d_x, *d_y;
    hipMalloc((void **)&d_csrRowPtrA, (m + 1) * sizeof(int));
    hipMalloc((void **)&d_csrColIndA, nnz * sizeof(int));
    hipMalloc((void **)&d_csrValA, nnz * sizeof(float));
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, m * sizeof(float));

    // Copy data to device
    hipMemcpy(d_csrRowPtrA, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrValA, csrValA, nnz * sizeof(float), hipMemcpyHostToDevice);

    // Input the vector
    printf("Enter the vector elements:\n");
    float *x = (float *)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++) {
        scanf("%f", &x[i]);
    }
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Perform the multiplication
    spmv_csr<<<(m + 255) / 256, 256>>>(d_csrRowPtrA, d_csrColIndA, d_csrValA, d_x, d_y, m);

    // Copy the result back to the host
    float *y = (float *)malloc(m * sizeof(float));
    hipMemcpy(y, d_y, m * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    printf("Result: ");
    for (int i = 0; i < m; i++) {
        printf("%f ", y[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_csrValA);
    hipFree(d_x);
    hipFree(d_y);
    free(A);
    free(csrRowPtrA);
    free(csrColIndA);
    free(csrValA);
    free(x);
    free(y);

    return 0;
}
