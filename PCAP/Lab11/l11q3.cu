#include "hip/hip_runtime.h"
// write a program in cuda to perform tiled 1d convolution 
// operation on the input array N of size width using the mask array ,  M of size mask_width, to produce the resultant array P of size width. 

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__global__ void tiledConvolution1D(float *N, float *M, float *P, int width, int mask_width, int padding) {
 
  __shared__ float s_Mask[BLOCK_SIZE];

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int i = idx + padding; 

  if (threadIdx.x < mask_width) {
    s_Mask[threadIdx.x] = M[threadIdx.x];
  }
  __syncthreads();

  if (i < width + padding) {
    float sum = 0;
    for (int j = 0; j < mask_width; j++) {
      int inputIndex = i - j;
      if (inputIndex >= 0) {
        sum += N[inputIndex] * s_Mask[j];
      }
    }
    P[idx] = sum;
  }
}

int main() {
  int width = 1024; /
  int mask_width = 5;
  int padding = mask_width / 2;

  float *h_N = (float*)malloc((width + 2 * padding) * sizeof(float));
  float *h_M = (float*)malloc(mask_width * sizeof(float));
  float *h_P = (float*)malloc(width * sizeof(float));

  for (int i = 0; i < width; i++) {
    h_N[i + padding] = rand() / (float)RAND_MAX; 
  }
  for (int i = 0; i < mask_width; i++) {
    h_M[i] = rand() / (float)RAND_MAX;
  }

  float *d_N, *d_M, *d_P;
  hipMalloc((void**)&d_N, (width + 2 * padding) * sizeof(float)); 
  hipMalloc((void**)&d_M, mask_width * sizeof(float));
  hipMalloc((void**)&d_P, width * sizeof(float));

  hipMemcpy(d_N, h_N, (width + 2 * padding) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_M, h_M, mask_width * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = BLOCK_SIZE;
  int gridSize = (width + blockSize - 1) / blockSize;

  tiledConvolution1D<<<gridSize, blockSize>>>(d_N, d_M, d_P, width, mask_width, padding);

  hipMemcpy(h_P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);

  printf("Final Output Matrix:\n");
  for (int i = 0; i < width; i++) {
    printf("%f ", h_P[i]);
  }
  printf("\n");

  free(h_N);
  free(h_M);
  free(h_P);
  hipFree(d_N);
  hipFree(d_M);
  hipFree(d_P);

  return 0;
}
