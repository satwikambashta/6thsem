// write a program in cuda to perform tiled 1d convolution 
// operation on the input array N of size width using the mask array ,  M of size mask_width, to produce the resultant array P of size width. 

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

// Kernel function for tiled 1D convolution
__global__ void tiledConvolution1D(float *N, float *M, float *P, int width, int mask_width) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < width) {
        float sum = 0;
        for (int i = 0; i < mask_width; i++) {
            int inputIndex = idx + i - mask_width / 2;
            if (inputIndex >= 0 && inputIndex < width) {
                sum += N[inputIndex] * M[i];
            }
        }
        P[idx] = sum;
    }
}

int main() {
    int width = 1024; // Size of the input array
    int mask_width = 5; // Size of the mask array

    float *h_N = (float*)malloc(width * sizeof(float));
    float *h_M = (float*)malloc(mask_width * sizeof(float));
    float *h_P = (float*)malloc(width * sizeof(float));

    // Initialize input and mask arrays
    for (int i = 0; i < width; i++) {
        h_N[i] = rand() / (float)RAND_MAX;
    }
    for (int i = 0; i < mask_width; i++) {
        h_M[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    float *d_N, *d_M, *d_P;
    hipMalloc((void**)&d_N, width * sizeof(float));
    hipMalloc((void**)&d_M, mask_width * sizeof(float));
    hipMalloc((void**)&d_P, width * sizeof(float));

    // Copy input and mask arrays to device
    hipMemcpy(d_N, h_N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, mask_width * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = BLOCK_SIZE;
    int gridSize = (width + blockSize - 1) / blockSize;

    // Perform tiled 1D convolution
    tiledConvolution1D<<<gridSize, blockSize>>>(d_N, d_M, d_P, width, mask_width);

    // Copy resultant array back to host
    hipMemcpy(h_P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    free(h_N);
    free(h_M);
    free(h_P);
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    return 0;
}
