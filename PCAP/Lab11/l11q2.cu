// write a program in cuda to improve the performance of 1d convolution using constant memory

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define FILTER_SIZE 5
#define BLOCK_SIZE 256

// Convolution kernel
__constant__ float d_Filter[FILTER_SIZE]; // Filter coefficients stored in constant memory

__global__ void convolution1D(float *d_Input, float *d_Output, int inputSize, int outputSize) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < outputSize) {
        float sum = 0;
        for (int i = 0; i < FILTER_SIZE; i++) {
            int inputIndex = idx + i - FILTER_SIZE / 2;
            if (inputIndex >= 0 && inputIndex < inputSize) {
                sum += d_Input[inputIndex] * d_Filter[i];
            }
        }
        d_Output[idx] = sum;
    }
}

int main() {
    int inputSize = 1024; // Size of the input array
    int outputSize = inputSize; // Size of the output array

    float *h_Input = (float*)malloc(inputSize * sizeof(float));
    float *h_Output = (float*)malloc(outputSize * sizeof(float));
    float h_Filter[FILTER_SIZE] = {0.1f, 0.2f, 0.3f, 0.2f, 0.1f}; // Example filter

    // Initialize input array
    for (int i = 0; i < inputSize; i++) {
        h_Input[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    float *d_Input, *d_Output;
    hipMalloc((void**)&d_Input, inputSize * sizeof(float));
    hipMalloc((void**)&d_Output, outputSize * sizeof(float));

    // Copy input and filter arrays to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_Filter), h_Filter, FILTER_SIZE * sizeof(float));
    hipMemcpy(d_Input, h_Input, inputSize * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int gridSize = (outputSize + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Perform convolution
    convolution1D<<<gridSize, BLOCK_SIZE>>>(d_Input, d_Output, inputSize, outputSize);

    // Copy output array back to host
    hipMemcpy(h_Output, d_Output, outputSize * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    free(h_Input);
    free(h_Output);
    hipFree(d_Input);
    hipFree(d_Output);

    return 0;
}
