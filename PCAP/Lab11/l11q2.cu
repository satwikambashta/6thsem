// write a program in cuda to improve the performance of 1d convolution using constant memory

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define FILTER_SIZE 5
#define BLOCK_SIZE 256

// Convolution kernel
__constant__ float d_Filter[FILTER_SIZE]; 

__global__ void convolution1D(float *d_Input, float *d_Output, int inputSize, int outputSize) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < outputSize) {
    float sum = 0;
    for (int i = 0; i < FILTER_SIZE; i++) {
      int inputIndex = idx + i - FILTER_SIZE / 2;
      if (inputIndex >= 0 && inputIndex < inputSize) {
        sum += d_Input[inputIndex] * d_Filter[i];
      }
    }
    d_Output[idx] = sum;
  }
}

int main() {
  int inputSize = 1024; 
  int outputSize = inputSize; 

  float *h_Input = (float*)malloc(inputSize * sizeof(float));
  float *h_Output = (float*)malloc(outputSize * sizeof(float));
  float filter[FILTER_SIZE] = {0.1f, 0.2f, 0.3f, 0.2f, 0.1f}; 

  for (int i = 0; i < inputSize; i++) {
    h_Input[i] = rand() / (float)RAND_MAX;
  }

  float *d_Input, *d_Output;
  hipMalloc((void**)&d_Input, inputSize * sizeof(float));
  hipMalloc((void**)&d_Output, outputSize * sizeof(float));

  hipMemcpy(d_Input, h_Input, inputSize * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(d_Filter), filter, FILTER_SIZE * sizeof(float));

  int gridSize = (outputSize + BLOCK_SIZE - 1) / BLOCK_SIZE;

  convolution1D<<<gridSize, BLOCK_SIZE>>>(d_Input, d_Output, inputSize, outputSize);

  hipMemcpy(h_Output, d_Output, outputSize * sizeof(float), hipMemcpyDeviceToHost);

  printf("Final Output Matrix:\n");
  for (int i = 0; i < outputSize; i++) {
    printf("%f ", h_Output[i]);
  }
  printf("\n");

  free(h_Input);
  free(h_Output);
  hipFree(d_Input);
  hipFree(d_Output);

  return 0;
}