// write a program in CUDA to perform tiled matrix multiplication using 2D grid and 2D block

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

// Kernel function for matrix multiplication
__global__ void matrixMul(float* A, float* B, float* C, int N) {
    __shared__ float tileA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tileB[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float tmp = 0;
    for (int m = 0; m < (N / BLOCK_SIZE); ++m) {
        tileA[threadIdx.y][threadIdx.x] = A[row * N + m * BLOCK_SIZE + threadIdx.x];
        tileB[threadIdx.y][threadIdx.x] = B[(m * BLOCK_SIZE + threadIdx.y) * N + col];
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            tmp += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }
    C[row * N + col] = tmp;
}

int main() {
    int N = 1024; // Matrix size
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocate memory on the host
    h_A = (float*)malloc(N * N * sizeof(float));
    h_B = (float*)malloc(N * N * sizeof(float));
    h_C = (float*)malloc(N * N * sizeof(float));

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    // Copy matrices to the device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Perform matrix multiplication
    dim3 dimGrid(N / BLOCK_SIZE, N / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to the host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
