//add two matrices by 
//a)each rowof resultant matrix to be computed by one thread
//b)each column if resultantmatrix is to be computed by one thread
//c)each element of resultant matrix to be computed by one thread

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void addMatricesRow(int *a, int *b, int *c, int m, int n)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        for (int col = 0; col < n; col++) {
            c[row * n + col] = a[row * n + col] + b[row * n + col];
        }
    }
}

__global__ void addMatricesColumn(int *a, int *b, int *c, int m, int n)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < n) {
        for (int row = 0; row < m; row++) {
            c[row * n + col] = a[row * n + col] + b[row * n + col];
        }
    }
}

__global__ void addMatricesElement(int *a, int *b, int *c, int m, int n)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < m && col < n) {
        c[row * n + col] = a[row * n + col] + b[row * n + col];
    }
}

int main(void)
{
    int m, n;
    printf("Enter the value of m: ");
    scanf("%d", &m);
    printf("Enter the value of n: ");
    scanf("%d", &n);

    int size = m * n;
    int *a = (int *)malloc(size * sizeof(int));
    int *b = (int *)malloc(size * sizeof(int));
    int *c = (int *)malloc(size * sizeof(int));

    printf("Enter matrix A:\n");
    for (int i = 0; i < size; i++)
        scanf("%d", &a[i]);

    printf("Enter matrix B:\n");
    for (int i = 0; i < size; i++)
        scanf("%d", &b[i]);

    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size * sizeof(int));
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_c, size * sizeof(int));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    int choice;
    printf("Enter 1 for rowwise, 2 for columnwise, 3 for elementwise: ");
    scanf("%d", &choice);
    switch(choice){
        case 1: 
            printf("ROW wISE: \n");
            addMatricesRow<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n);
            break;
        case 2: 
            printf("COLUMN WISE; \n");
            addMatricesColumn<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n);
            break;
        case 3: 
            addMatricesElement<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n);
            break;
        default:
            printf("Incorrect input. defaulting to elemenmtwise\n");
    }

    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result matrix is:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d\t", c[i * n + j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
