//multiply two matrices for specifications
//a)each rowof resultant matrix to be computed by one thread
//b)each column if resultantmatrix is to be computed by one thread
//c)each element of resultant matrix to be computed by one thread

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void multiplyMatricesRow(int *a, int *b, int *c, int m, int n, int p)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        for (int col = 0; col < p; col++) {
            int sum = 0;
            for (int k = 0; k < n; k++) {
                sum += a[row * n + k] * b[k * p + col];
            }
            c[row * p + col] = sum;
        }
    }
}

__global__ void multiplyMatricesColumn(int *a, int *b, int *c, int m, int n, int p)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < p) {
        for (int row = 0; row < m; row++) {
            int sum = 0;
            for (int k = 0; k < n; k++) {
                sum += a[row * n + k] * b[k * p + col];
            }
            c[row * p + col] = sum;
        }
    }
}

__global__ void multiplyMatricesElement(int *a, int *b, int *c, int m, int n, int p)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < m && col < p) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * p + col];
        }
        c[row * p + col] = sum;
    }
}

int main(void)
{
    int m, n, p;
    printf("Enter the value of m: ");
    scanf("%d", &m);
    printf("Enter the value of n: ");
    scanf("%d", &n);
    printf("Enter the value of p: ");
    scanf("%d", &p);

    int sizeA = m * n;
    int sizeB = n * p;
    int sizeC = m * p;

    int *a = (int *)malloc(sizeA * sizeof(int));
    int *b = (int *)malloc(sizeB * sizeof(int));
    int *c = (int *)malloc(sizeC * sizeof(int));

    printf("Enter matrix A (%d x %d):\n", m, n);
    for (int i = 0; i < sizeA; i++)
        scanf("%d", &a[i]);

    printf("Enter matrix B (%d x %d):\n", n, p);
    for (int i = 0; i < sizeB; i++)
        scanf("%d", &b[i]);

    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, sizeA * sizeof(int));
    hipMalloc((void **)&d_b, sizeB * sizeof(int));
    hipMalloc((void **)&d_c, sizeC * sizeof(int));

    hipMemcpy(d_a, a, sizeA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeB * sizeof(int), hipMemcpyHostToDevice);

    int choice;
    printf("Choose the approach (1: Row-wise, 2: Column-wise, 3: Element-wise): ");
    scanf("%d", &choice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((p + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    switch (choice) {
        case 1:
            multiplyMatricesRow<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n, p);
            break;
        case 2:
            multiplyMatricesColumn<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n, p);
            break;
        case 3:
            multiplyMatricesElement<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n, p);
            break;
        default:
            printf("Invalid choice.\n");
            return 1;
    }

    hipMemcpy(c, d_c, sizeC * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result matrix is:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < p; j++) {
            printf("%d\t", c[i * p + j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
