// addd two vectors of length N by keeping the number of threADS PER BLOCK AS 256 AND VARY the number of
//  blocks to handle N elements 


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void vectorAdd_q2(float *A, float *B,float *C,int n)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;

	if(tid<n)
		C[tid]=A[tid]+B[tid];
}

int main(void) 
{
	int n;

	printf("Enter number of elements:\t");
	scanf("%d",&n);

	float *h_A,*h_B,*h_C;
	float *d_A, *d_B, *d_C;
	
	int size = sizeof(float)*n;

	h_A=(float*)malloc(sizeof(size));
	h_B=(float*)malloc(sizeof(size));
	h_C=(float*)malloc(sizeof(size));

	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	printf("Enter vector 1:\n");
	for(int i=0;i<n;i++)
		scanf("%f",&h_A[i]);

	printf("Enter vector 2:\n");
	for(int i=0;i<n;i++)
		scanf("%f",&h_B[i]);

    printf("\nOriginal array 1: \n");
    for(int i=0;i<n;i++)
		printf("\t %f",h_A[i]);
    printf("\nOriginal array 2: \n");
    for(int i=0;i<n;i++)
		printf("\t %f",h_B[i]);



	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("Resultant Vector Q1_B:\n");
	for(int i=0;i<n;i++)
		printf("%f\t",h_C[i]);

	printf("\n");

	dim3 dimGrid_q2(ceil(n/256.0),1,1);
	dim3 dimBlock_q2(256,1,1);

	vectorAdd_q2<<<dimGrid_q2,dimBlock_q2>>>(d_A,d_B,d_C,n);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("Resultant Vector Q2:\n");
	for(int i=0;i<n;i++)
		printf("%f\t",h_C[i]);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}
